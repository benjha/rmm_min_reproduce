
#include <algorithm>
#include <iostream>
#include <numeric>
#include <array>
#include <vector>
#include <stdlib.h>
#include <random>
#include <thread>

#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system/cuda/execution_policy.h>
#include "thrust_rmm_allocator.h"


#include "cClipping.h"

typedef rmm::device_vector<float>::iterator IterFloat;
typedef rmm::device_vector<int>::iterator IterInt;

int main(int argc, char *argv[]){

  size_t sx, sy, sz;
  
  std::vector<float> pos;
  rmm::device_vector<float> d_pos;

  // Types of allocations:
	// CudaDefaultAllocation
	// PoolAllocation
	// CudaManagedMemory

  rmmOptions_t options{static_cast<rmmAllocationMode_t>(PoolAllocation | CudaManagedMemory), 0, true};
  rmmInitialize(&options);
    
  hipStream_t stream;
	if (hipStreamCreate (&stream) !=  hipSuccess){
    std::cout<< "stream error";
  }

  if (argc < 4){
    std::cout << "Usage: clipping x_size y_size z_size" << std::endl;
    return 1;
  }
	sx = std::stoll (std::string(argv[1]));
	sy = std::stoll (std::string(argv[2]));
	sz = std::stoll (std::string(argv[3]));
   
  initDataset(&pos, sx, sy, sz);
  d_pos = pos;
  float normal[3], d = 0.0f;
  normal[0] = 0.5f;
  normal[1] = 0.5f;
  normal[2] = 0.5f;
  
  plane_clippingPDBver2 clip	(normal, d);

	strided_range<IterFloat> X		( d_pos.begin()  , d_pos.end(), 4);
	strided_range<IterFloat> Y		( d_pos.begin()+1, d_pos.end(), 4);
	strided_range<IterFloat> Z		( d_pos.begin()+2, d_pos.end(), 4);
  strided_range<IterFloat> W		( d_pos.begin()+3, d_pos.end(), 4);
  
	size_t new_size = thrust::remove_if(rmm::exec_policy(stream)->on(stream), thrust::make_zip_iterator ( thrust::make_tuple( X.begin(), Y.begin(), Z.begin(), W.begin() )),
			   	   	   	   	   	   	   	 thrust::make_zip_iterator ( thrust::make_tuple( X.end(),Y.end(), Z.end(), W.end() )),
			   	   	   	   	   	   	   	 clip )
                                      - thrust::make_zip_iterator(thrust::make_tuple(X.begin(), Y.begin(), Z.begin(), W.begin()));
  return 0;
}
